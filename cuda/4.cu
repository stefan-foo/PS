#include "hip/hip_runtime.h"
%%cu
#include <vector>
#include <iostream>
#define N 125
#define BLOCK_DIM 256
#define MAX_GRID_SIZE 256
#define SHMEM_SIZE 4 * BLOCK_DIM

__global__ void componentProduct(int* a, int* b, int* out, int len) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;

  while (tid < len) {
    out[tid] = a[tid] * b[tid];
    tid += blockDim.x * gridDim.x;
  }
}

__global__ void reduceSum(int* v, int* v_r, int len) {
  __shared__ int partial_sum[SHMEM_SIZE];

  int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;

  if (i < len) {
      partial_sum[threadIdx.x] = v[i];
      if (i + blockDim.x < len) {
          partial_sum[threadIdx.x] += v[i + blockDim.x];
      }
  } else {
      partial_sum[threadIdx.x] = 0;
  }

  __syncthreads();

  for (int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (threadIdx.x < s) 
      partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];

    __syncthreads();
  }

  if (threadIdx.x == 0) {
    v_r[blockIdx.x] = partial_sum[0];
  }
}

int main(void)
{
  std::vector<int> a(N);
  std::vector<int> b(N);
  
  for (int i = 0; i < N; i++) {
    a[i] = 1;
    b[i] = i + 1;
  }

  int* d_a, *d_b, *d_c_s;
  hipMalloc((void**)&d_a, sizeof(int) * N);
  hipMalloc((void**)&d_b, sizeof(int) * N);
  hipMalloc((void**)&d_c_s, sizeof(int) * N);

  hipMemcpy(d_a, a.data(), sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b.data(), sizeof(int) * N, hipMemcpyHostToDevice);

  componentProduct<<<min(MAX_GRID_SIZE, (N + BLOCK_DIM - 1) / BLOCK_DIM), BLOCK_DIM>>>(d_a, d_b, d_c_s, N); 

  int* d_v_r;
  hipMalloc((void**)&d_v_r, sizeof(int) * N);
 
  int GRID_DIM = (N + BLOCK_DIM * 2 - 1) / (BLOCK_DIM * 2);
  reduceSum<<<GRID_DIM, BLOCK_DIM>>>(d_c_s, d_v_r, N);
  reduceSum<<<1, GRID_DIM>>>(d_v_r, d_v_r, GRID_DIM);

  int result;
  hipMemcpy(&result, d_v_r, sizeof(int), hipMemcpyDeviceToHost);

  std::cout << result << std::endl;

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c_s);
  hipFree(d_v_r);
}